#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <ctime>
#include <vector>
#include <exception>
#include <unistd.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <cutf/device.hpp>
#include <cutf/nvml.hpp>

void parse_params(unsigned &time_interval, std::string& output_file_name, int& run_command_head, int argc, char** argv) {
	run_command_head = 1;
	output_file_name = "gpu.log";
	time_interval = 1;
	for (int i = 1; i < argc;) {
		if (std::string(argv[i]) == "-i") {
			if (i + 1 >= argc) {
				throw std::runtime_error("The value of `-i` is not provided");
			}
			time_interval = std::stoul(argv[i+1]);
			i += 2;
		} else if (std::string(argv[i]) == "-o") {
			if (i + 1 >= argc) {
				throw std::runtime_error("The value of `-o` is not provided");
			}
			output_file_name = argv[i+1];
			i += 2;
		} else if (std::string(argv[i]) == "-h") {
			time_interval = 0; // This means that this execution is invalid and exits with printing help messages.
		} else {
			run_command_head = i;
			return;
		}
	}
}

void print_help_message(const char* const program_name) {
	std::printf("/*** GPU Logger ***/\n");
	std::printf("\n");
	std::printf("// Usage\n");
	std::printf("%s [-i interval(s){default=1}] [-o output_file_name{default=gpu.csv}] target_command\n", program_name);
}

namespace process {
constexpr char running = 'R';
constexpr char end     = 'E';
} // namespace process

int main(int argc, char** argv) {
	std::string output_file_name;
	unsigned time_interval;
	int run_command_head;

	parse_params(time_interval, output_file_name, run_command_head, argc, argv);

	if (time_interval < 1 || argc <= 1) {
		print_help_message(argv[0]);
		return 1;
	}

	const auto fd = shm_open("/gpu_logger_smem", O_CREAT | O_RDWR, 0666);
	ftruncate(fd, 1);
	const auto semaphore = static_cast<char*>(mmap(nullptr, 1, PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0));
	*semaphore = process::running;

	const auto pid = fork();
	if (pid == 0) {
		std::ofstream ofs(output_file_name);
		CUTF_CHECK_ERROR(nvmlInit());
		const auto num_devices = cutf::device::get_num_devices();

		// Output csv header
		ofs << "index,date,";
		for (unsigned gpu_id = 0; gpu_id < num_devices; gpu_id++) {
			ofs << "gpu" << gpu_id << "_temp,";
			ofs << "gpu" << gpu_id << "_power,";
			ofs << "gpu" << gpu_id << "_memory_used,";
		}
		ofs << "\n";
		ofs.close();

		// Output log
		unsigned count = 0;
		while ((*semaphore) == process::running) {
			std::ofstream ofs(output_file_name, std::ios::app);
			ofs << (count++) << ","
				<< std::time(nullptr) << ",";
			for (unsigned gpu_id = 0; gpu_id < num_devices; gpu_id++) {
				nvmlDevice_t device;
				CUTF_CHECK_ERROR(nvmlDeviceGetHandleByIndex(gpu_id, &device));

				unsigned int temperature;
				CUTF_CHECK_ERROR(nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperature));
				nvmlMemory_t memory;
				CUTF_CHECK_ERROR(nvmlDeviceGetMemoryInfo(device, &memory));
				unsigned int power;
				CUTF_CHECK_ERROR(nvmlDeviceGetPowerUsage(device, &power));

				ofs << temperature << ","
					<< (power / 1000.0) << ","
					<< memory.used << ",";
			}
			ofs << "\n";
			ofs.close();
			sleep(time_interval);
		}

		CUTF_CHECK_ERROR(nvmlShutdown());
	} else {
		const auto cmd = argv[run_command_head];
		std::vector<char*> cmd_args(argc - run_command_head + 1);	
		for (int i = run_command_head, v = 0; i < argc; i++, v++) {
			cmd_args[v] = argv[i];
		}
		cmd_args[cmd_args.size() - 1] = nullptr;

		execvp(cmd, cmd_args.data());
		*semaphore = process::end;
		exit(0);
	}
}
