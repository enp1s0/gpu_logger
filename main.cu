#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <ctime>
#include <chrono>
#include <vector>
#include <exception>
#include <unistd.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/wait.h>
#include <fcntl.h>
#include <cutf/device.hpp>
#include <cutf/nvml.hpp>

void parse_params(unsigned &time_interval, std::string& output_file_name, int& run_command_head, int argc, char** argv) {
	run_command_head = 1;
	output_file_name = "gpu.csv";
	time_interval = 1;
	for (int i = 1; i < argc;) {
		if (std::string(argv[i]) == "-i") {
			if (i + 1 >= argc) {
				throw std::runtime_error("The value of `-i` is not provided");
			}
			time_interval = std::stoul(argv[i+1]);
			i += 2;
		} else if (std::string(argv[i]) == "-o") {
			if (i + 1 >= argc) {
				throw std::runtime_error("The value of `-o` is not provided");
			}
			output_file_name = argv[i+1];
			i += 2;
		} else if (std::string(argv[i]) == "-h") {
			time_interval = 0; // This means that this execution is invalid and exits with printing help messages.
		} else {
			run_command_head = i;
			return;
		}
	}
}

void print_help_message(const char* const program_name) {
	std::printf("/*** GPU Logger ***/\n");
	std::printf("\n");
	std::printf("// Usage\n");
	std::printf("%s [-i interval(ms){default=1000}] [-o output_file_name{default=gpu.csv}] target_command\n", program_name);
}

namespace process {
constexpr char running = 'R';
constexpr char end     = 'E';
} // namespace process

int main(int argc, char** argv) {
	std::string output_file_name;
	unsigned time_interval;
	int run_command_head;

	parse_params(time_interval, output_file_name, run_command_head, argc, argv);

	if (time_interval < 1 || argc <= 1) {
		print_help_message(argv[0]);
		return 1;
	}

	const auto fd = shm_open("/gpu_logger_smem", O_CREAT | O_RDWR, 0666);
	ftruncate(fd, 1);
	const auto semaphore = static_cast<char*>(mmap(nullptr, 1, PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0));
	*semaphore = process::running;

	const auto pid = fork();
	if (pid == 0) {
		std::ofstream ofs(output_file_name);
		CUTF_CHECK_ERROR(nvmlInit());
		const auto num_devices = cutf::device::get_num_devices();

		// Output csv header
		ofs << "index,date,elapsed_time,";
		for (unsigned gpu_id = 0; gpu_id < num_devices; gpu_id++) {
			ofs << "gpu" << gpu_id << "_temp,";
			ofs << "gpu" << gpu_id << "_power,";
			ofs << "gpu" << gpu_id << "_memory_used,";
		}
		ofs << "\n";
		ofs.close();

		// Output log
		unsigned count = 0;
		const auto start_clock = std::chrono::high_resolution_clock::now();
		while ((*semaphore) == process::running) {
			std::ofstream ofs(output_file_name, std::ios::app);
			ofs << (count++) << ","
				<< std::time(nullptr) << ",";
			const auto end_clock = std::chrono::high_resolution_clock::now();
			ofs << std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() << ",";
			for (unsigned gpu_id = 0; gpu_id < num_devices; gpu_id++) {
				nvmlDevice_t device;
				CUTF_CHECK_ERROR(nvmlDeviceGetHandleByIndex(gpu_id, &device));

				unsigned int temperature;
				CUTF_CHECK_ERROR(nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperature));
				nvmlMemory_t memory;
				CUTF_CHECK_ERROR(nvmlDeviceGetMemoryInfo(device, &memory));
				unsigned int power;
				CUTF_CHECK_ERROR(nvmlDeviceGetPowerUsage(device, &power));

				ofs << temperature << ","
					<< (power / 1000.0) << ","
					<< memory.used << ",";
			}
			ofs << "\n";
			ofs.close();
			usleep(time_interval * 1000);
		}

		CUTF_CHECK_ERROR(nvmlShutdown());
		exit(0);
	} else {
		const auto cmd = argv[run_command_head];
		std::vector<char*> cmd_args(argc - run_command_head + 1);	
		for (int i = run_command_head, v = 0; i < argc; i++, v++) {
			cmd_args[v] = argv[i];
		}
		cmd_args[cmd_args.size() - 1] = nullptr;

		const auto command_pid = fork();
		if (command_pid == 0) {
			execvp(cmd, cmd_args.data());
			exit(0);
		} else {
			wait(nullptr);
			*semaphore = process::end;
		}
	}
}
