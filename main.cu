#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <ctime>
#include <vector>
#include <exception>
#include <unistd.h>
#include <cutf/device.hpp>
#include <cutf/nvml.hpp>

void parse_params(unsigned &time_interval, std::string& output_file_name, int& run_command_head, int argc, char** argv) {
	run_command_head = 1;
	output_file_name = "gpu.log";
	time_interval = 1;
	for (int i = 1; i < argc; i++) {
		if (std::string(argv[i]) == "-i") {
			if (i + 1 >= argc) {
				throw std::runtime_error("The value of `-i` is not provided");
			}
			time_interval = std::stoul(argv[i+1]);
			i += 2;
		} else if (std::string(argv[i]) == "-i") {
			if (i + 1 >= argc) {
				throw std::runtime_error("The value of `-o` is not provided");
			}
			output_file_name = argv[i+1];
			i += 2;
		} else {
			run_command_head = i;
			return;
		}
	}
}

int main(int argc, char** argv) {
	std::string output_file_name;
	unsigned time_interval;
	int run_command_head;

	parse_params(time_interval, output_file_name, run_command_head, argc, argv);

	int pid = fork();
	if (pid != 0) {
		std::ofstream ofs(output_file_name);
		CUTF_CHECK_ERROR(nvmlInit());
		const auto num_devices = cutf::device::get_num_devices();

		// Output csv header
		ofs << "index,date,";
		for (unsigned gpu_id = 0; gpu_id < num_devices; gpu_id++) {
			ofs << "gpu" << gpu_id << "_temp,";
			ofs << "gpu" << gpu_id << "_power,";
			ofs << "gpu" << gpu_id << "_memory,";
		}
		ofs << "\n";
		ofs.close();

		// Output log
		unsigned count = 0;
		while (1) {
			std::ofstream ofs(output_file_name, std::ios::app);
			ofs << std::time(nullptr) << ","
				<< (count++) << ",";
			for (unsigned gpu_id = 0; gpu_id < num_devices; gpu_id++) {
				nvmlDevice_t device;
				CUTF_CHECK_ERROR(nvmlDeviceGetHandleByIndex(gpu_id, &device));

				unsigned int temperature;
				CUTF_CHECK_ERROR(nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temperature));
				unsigned int memory;
				CUTF_CHECK_ERROR(nvmlDeviceGetMemoryInfo(device, memory));
				unsigned int power;
				CUTF_CHECK_ERROR(nvmlDeviceGetPowerUsage(device, power));

				ofs << temperature << ","
					<< power << ","
					<< memory << ",";
			}
			ofs.close();
			sleep(time_interval * 1000);
		}

		CUTF_CHECK_ERROR(nvmlShutdown());
	} else {
		const auto cmd = argv[run_command_head];
		std::vector<char*> cmd_args(argc - run_command_head + 1);	
		for (int i = 0, v = 0; i < run_command_head; i++, v++) {
			cmd_args[v] = argv[i];
		}
		cmd_args[cmd_args.size() - 1] = nullptr;

		execvp(cmd, cmd_args.data());
		exit(0);
	}
}
